#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 256  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define N 256  // Number of columns in B and C
#define BLOCK_SIZE 32

// CPU matrix multiplication
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int l = 0; l < k; l++) {
                sum += A[i * k + l] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// CUDA kernel for matrix multiplication
__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 5; i++) {
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    float cpu_time_elapsed = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
    }
    hipEventRecord(stop);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_time_elapsed, start, stop);
    printf("CPU time elapsed: %f ms\n", cpu_time_elapsed / 100.0);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    float gpu_time_elapsed = 0.0f;
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);
    for (int i = 0; i < 100; i++) {
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(start_gpu);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&gpu_time_elapsed, start_gpu, stop_gpu);
    printf("GPU time elapsed: %f ms\n", gpu_time_elapsed / 100.0);
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    // Print results
    printf("CPU average time: %f microseconds\n", (cpu_time_elapsed * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_time_elapsed * 1e6f));
    printf("Speedup: %fx\n", cpu_time_elapsed / gpu_time_elapsed);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}